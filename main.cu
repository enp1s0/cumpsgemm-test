#include <cumpsgemm/cumpsgemm.hpp>
#include <sstream>
#include <stdexcept>

#ifndef CHECK_ERROR
#define CHECK_ERROR(status) cuda_check(status, __FILE__, __LINE__, __func__)
#endif

namespace {
inline void cuda_check(hipError_t error, const std::string filename, const std::size_t line, const std::string funcname){
	if(error != hipSuccess){
		std::stringstream ss;
		ss<< hipGetErrorString( error );
		ss<<" ["<<filename<<":"<<line<<" in "<<funcname<<"]";
		throw std::runtime_error(ss.str());
	}
}
}

constexpr std::size_t N = 2048;

int main() {
	float *a_ptr, *b_ptr, *c_ptr;
	CHECK_ERROR(hipMalloc(&a_ptr, sizeof(float) * N * N));
	CHECK_ERROR(hipMalloc(&b_ptr, sizeof(float) * N * N));
	CHECK_ERROR(hipMalloc(&c_ptr, sizeof(float) * N * N));

	cumpsgemm::handle_t cumpsgemm_handle;
	cumpsgemm::create(cumpsgemm_handle);
	//cumpsgemm::set_stream(cumpsgemm_handle, hip_stream);

	float alpha = 1.f, beta = 0.f;
	cumpsgemm::gemm(
			cumpsgemm_handle,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			N, N, N,
			&alpha,
			a_ptr, N,
			b_ptr, N,
			&beta,
			c_ptr, N,
			CUMPSGEMM_TF32TCEC
			);

	CHECK_ERROR(hipFree(a_ptr));
	CHECK_ERROR(hipFree(b_ptr));
	CHECK_ERROR(hipFree(c_ptr));
}
